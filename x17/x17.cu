#include "hip/hip_runtime.h"
/**
 * X17R algorithm (X17 with Randomized chain order)
 *
 * tpruvot 2018 - GPL code
 * Copyright (c) 2020 XpsCommunity team
 */

#include <stdio.h>
#include <memory.h>
#include <unistd.h>

extern "C" {
#include "sph/sph_blake.h"
#include "sph/sph_bmw.h"
#include "sph/sph_groestl.h"
#include "sph/sph_skein.h"
#include "sph/sph_jh.h"
#include "sph/sph_keccak.h"

#include "sph/sph_luffa.h"
#include "sph/sph_cubehash.h"
#include "sph/sph_shavite.h"
#include "sph/sph_simd.h"
#include "sph/sph_echo.h"

#include "sph/sph_hamsi.h"
#include "sph/sph_fugue.h"
#include "sph/sph_shabal.h"
#include "sph/sph_whirlpool.h"
#include "sph/sph_sha2.h"
#include "sph/sph_haval.h"
}

#include "miner.h"
#include "cuda_helper.h"
#include "cuda_x17.h"

static uint32_t *d_hash[MAX_GPUS];

enum Algo {
	BLAKE = 0,
	BMW,
	GROESTL,
	JH,
	KECCAK,
	SKEIN,
	LUFFA,
	CUBEHASH,
	SHAVITE,
	SIMD,
	ECHO,
	HAMSI,
	FUGUE,
	SHABAL,
	WHIRLPOOL,
	SHA512,
	HAVAL,
	HASH_FUNC_COUNT
};

static const char* algo_strings[] = {
	"blake",
	"bmw512",
	"groestl",
	"jh512",
	"keccak",
	"skein",
	"luffa",
	"cube",
	"shavite",
	"simd",
	"echo",
	"hamsi",
	"fugue",
	"shabal",
	"whirlpool",
	"sha512",
	"haval256",
	NULL
};

static __thread uint32_t s_ntime = UINT32_MAX;
static __thread bool s_implemented = false;
static __thread char hashOrder[HASH_FUNC_COUNT + 1] = { 0 };
static __thread bool x17_context_init = false;

static void getAlgoString(const uint32_t* prevblock, char *output)
{
	char *sptr = output;
	uint8_t* data = (uint8_t*)prevblock;

	for (uint8_t j = 0; j < HASH_FUNC_COUNT; j++) {
		//uint8_t b = (15 - j) >> 1; // 16 ascii hex chars, reversed
		//uint8_t algoDigit = (j & 1) ? data[b] & 0xF : data[b] >> 4;
		uint8_t algoDigit = data[j] % HASH_FUNC_COUNT;
		if (algoDigit >= 10)
			sprintf(sptr, "%c", 'A' + (algoDigit - 10));
		else
			sprintf(sptr, "%u", (uint32_t) algoDigit);
		sptr++;
	}
	*sptr = '\0';
}


struct x17_contexts
{
	sph_blake512_context ctx_blake;
	sph_bmw512_context ctx_bmw;
	sph_groestl512_context ctx_groestl;
	sph_jh512_context ctx_jh;
	sph_keccak512_context ctx_keccak;
	sph_skein512_context ctx_skein;
	sph_luffa512_context ctx_luffa;
	sph_cubehash512_context ctx_cubehash;
	sph_shavite512_context ctx_shavite;
	sph_simd512_context ctx_simd;
	sph_echo512_context ctx_echo;
	sph_hamsi512_context ctx_hamsi;
	sph_fugue512_context ctx_fugue;
	sph_shabal512_context ctx_shabal;
	sph_whirlpool_context ctx_whirlpool;
	sph_sha512_context ctx_sha512;
	sph_haval256_5_context ctx_haval;
};

static __thread x17_contexts base_contexts;

static void init_contexts(x17_contexts *ctx)
{
	sph_blake512_init(&ctx->ctx_blake);
	sph_bmw512_init(&ctx->ctx_bmw);
	sph_groestl512_init(&ctx->ctx_groestl);
	sph_skein512_init(&ctx->ctx_skein);
	sph_jh512_init(&ctx->ctx_jh);
	sph_keccak512_init(&ctx->ctx_keccak);
	sph_luffa512_init(&ctx->ctx_luffa);
	sph_cubehash512_init(&ctx->ctx_cubehash);
	sph_shavite512_init(&ctx->ctx_shavite);
	sph_simd512_init(&ctx->ctx_simd);
	sph_echo512_init(&ctx->ctx_echo);
	sph_hamsi512_init(&ctx->ctx_hamsi);
	sph_fugue512_init(&ctx->ctx_fugue);
	sph_shabal512_init(&ctx->ctx_shabal);
	sph_whirlpool_init(&ctx->ctx_whirlpool);
	sph_sha512_init(&ctx->ctx_sha512);
	sph_haval256_5_init(&ctx->ctx_haval);
}


// X17R CPU Hash (Validation)
extern "C" void x17_hash(void *output, const void *input)
{
	unsigned char _ALIGN(64) hash[128];

	

	void *in = (void*) input;
	int size = 80;

	uint32_t *in32 = (uint32_t*) input;
	getAlgoString(&in32[1], hashOrder);
	//applog(LOG_INFO, "hashOrder %s ", hashOrder);
	memset(&hash, 0, 128);
	x17_contexts ctx;
	if (!x17_context_init) {
		init_contexts(&base_contexts);
		x17_context_init = true;
	}
	memcpy(&ctx, &base_contexts, sizeof(x17_contexts));
	

	
	sph_blake512(&ctx.ctx_blake, in, size);
	sph_blake512_close(&ctx.ctx_blake, hash);

	size = 64;
	
	sph_bmw512(&ctx.ctx_bmw, hash, size);
	sph_bmw512_close(&ctx.ctx_bmw, hash);

	
	sph_groestl512(&ctx.ctx_groestl, hash, size);
	sph_groestl512_close(&ctx.ctx_groestl, hash);

	
	sph_skein512(&ctx.ctx_skein, hash, size);
	sph_skein512_close(&ctx.ctx_skein, hash);

	
	sph_jh512(&ctx.ctx_jh, hash, size);
	sph_jh512_close(&ctx.ctx_jh, hash);

	
	sph_keccak512(&ctx.ctx_keccak, hash, size);
	sph_keccak512_close(&ctx.ctx_keccak, hash);

	
	sph_luffa512(&ctx.ctx_luffa, hash, size);
	sph_luffa512_close(&ctx.ctx_luffa, hash);

	
	sph_cubehash512(&ctx.ctx_cubehash, hash, size);
	sph_cubehash512_close(&ctx.ctx_cubehash, hash);


	
	sph_shavite512(&ctx.ctx_shavite, hash, size);
	sph_shavite512_close(&ctx.ctx_shavite, hash);

	
	sph_simd512(&ctx.ctx_simd, hash, size);
	sph_simd512_close(&ctx.ctx_simd, hash);

	
	sph_echo512(&ctx.ctx_echo, hash, size);
	sph_echo512_close(&ctx.ctx_echo, hash);

	
	sph_hamsi512(&ctx.ctx_hamsi, hash, size);
	sph_hamsi512_close(&ctx.ctx_hamsi, hash);


	
	sph_fugue512(&ctx.ctx_fugue, hash, size);
	sph_fugue512_close(&ctx.ctx_fugue, hash);


	
	sph_shabal512(&ctx.ctx_shabal, hash, size);
	sph_shabal512_close(&ctx.ctx_shabal, hash);

	
	sph_whirlpool(&ctx.ctx_whirlpool, hash, size);
	sph_whirlpool_close(&ctx.ctx_whirlpool, hash);


	
	sph_sha512(&ctx.ctx_sha512,(const void*) hash, size);
	sph_sha512_close(&ctx.ctx_sha512,(void*) hash);

	
	sph_haval256_5(&ctx.ctx_haval, (const void*)hash, size);
	sph_haval256_5_close(&ctx.ctx_haval, hash);

	memcpy(output, hash, 32);
}

static bool init[MAX_GPUS] = { 0 };

//#define _DEBUG
#define _DEBUG_PREFIX "x17-"
#include "cuda_debug.cuh"

//static int algo80_tests[HASH_FUNC_COUNT] = { 0 };
//static int algo64_tests[HASH_FUNC_COUNT] = { 0 };
static int algo80_fails[HASH_FUNC_COUNT] = { 0 };

extern "C" int scanhash_x17(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];
	const int dev_id = device_map[thr_id];
	
	//int intensity = (device_sm[dev_id] > 500 && !is_windows()) ? 20 : 19;
	//if (strstr(device_name[dev_id], "GTX 1080")) intensity = 20;
	//uint32_t throughput = cuda_default_throughput(thr_id, 1U << intensity);
	//if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	uint32_t default_throughput = 1 << 20;
	if ((strstr(device_name[dev_id], "1050")))
	{
		default_throughput = 1 << 20;
	}
	else if ((strstr(device_name[dev_id], "950")))
	{
		default_throughput = 1 << 20;
	}
	else if ((strstr(device_name[dev_id], "960")))
	{
		default_throughput = 1 << 20;
	}
	else if ((strstr(device_name[dev_id], "750")))
	{
		default_throughput = 1 << 20;
	}
	else if ((strstr(device_name[dev_id], "1060")) || (strstr(device_name[dev_id], "P106")))
	{
		default_throughput = (1 << 21);
	}
	else if ((strstr(device_name[dev_id], "970") || (strstr(device_name[dev_id], "980"))))
	{
		default_throughput = (1 << 21);
	}
	else if ((strstr(device_name[dev_id], "166")) || (strstr(device_name[dev_id], "20")))
	{
		default_throughput = (1 << 21);
	}
	else if (strstr(device_name[dev_id], "1070") || (strstr(device_name[dev_id], "P104")))
	{
		default_throughput = (1 << 21);
	}
	else if ((strstr(device_name[dev_id], "1080 Ti")) || (strstr(device_name[dev_id], "1080")) || (strstr(device_name[dev_id], "P102")))
	{
		default_throughput = (1 << 21);
	}
	uint32_t throughput = cuda_default_throughput(thr_id, default_throughput);
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
		}
		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		quark_blake512_cpu_init(thr_id, throughput);
		quark_bmw512_cpu_init(thr_id, throughput);
		quark_groestl512_cpu_init(thr_id, throughput);
		quark_skein512_cpu_init(thr_id, throughput);
		quark_jh512_cpu_init(thr_id, throughput);
		quark_keccak512_cpu_init(thr_id, throughput);
		qubit_luffa512_cpu_init(thr_id, throughput);
		x11_luffa512_cpu_init(thr_id, throughput); // 64
		x11_shavite512_cpu_init(thr_id, throughput);
		x11_simd512_cpu_init(thr_id, throughput); // 64
		x11_echo512_cpu_init(thr_id, throughput);
		x16_echo512_cuda_init(thr_id, throughput);
		x13_hamsi512_cpu_init(thr_id, throughput);
		x13_fugue512_cpu_init(thr_id, throughput);
		x16_fugue512_cpu_init(thr_id, throughput);
		x14_shabal512_cpu_init(thr_id, throughput);
		x15_whirlpool_cpu_init(thr_id, throughput, 0);
		x16_whirlpool512_init(thr_id, throughput);
		x17_sha512_cpu_init(thr_id, throughput);
		x17_haval256_cpu_init(thr_id, throughput);

		CUDA_CALL_OR_RET_X(hipMalloc(&d_hash[thr_id], (size_t) 64 * throughput), 0);

		cuda_check_cpu_init(thr_id, throughput);

		init[thr_id] = true;
	}

	if (opt_benchmark) {
		//((uint32_t*)ptarget)[7] = 0x003f;
		//((uint8_t*)pdata)[8] = 0x90; // hashOrder[0] = '9'; for simd 80 + blake512 64
		//((uint8_t*)pdata)[8] = 0xA0; // hashOrder[0] = 'A'; for echo 80 + blake512 64
		//((uint8_t*)pdata)[8] = 0xB0; // hashOrder[0] = 'B'; for hamsi 80 + blake512 64
		//((uint8_t*)pdata)[8] = 0xC0; // hashOrder[0] = 'C'; for fugue 80 + blake512 64
		//((uint8_t*)pdata)[8] = 0xE0; // hashOrder[0] = 'E'; for whirlpool 80 + blake512 64
	}
	uint32_t _ALIGN(64) endiandata[20];

	for (int k=0; k < 20; k++)
		endiandata[k] = pdata[k];

	//char endiandata_str[161];
	//memset(endiandata_str, 0x0, sizeof(endiandata_str));
	//for (int k = 0; k < 80; k++)
	//	sprintf(endiandata_str + 2 * k, "%02x", ((uint8_t*)endiandata)[k]);
	//printf("%s\n", endiandata_str);

	uint32_t ntime = swab32(pdata[0]);
	if (s_ntime != ntime) {
		getAlgoString(&endiandata[1], hashOrder);
		//applog(LOG_INFO, "hashOrder %s ", hashOrder);
		s_ntime = ntime;
		s_implemented = true;
		if (opt_debug && !thr_id) applog(LOG_DEBUG, "hash order %s (%08x)", hashOrder, ntime);
	}

	if (!s_implemented) {
		sleep(1);
		return -1;
	}

	cuda_check_cpu_setTarget(ptarget);

	
	quark_blake512_cpu_setBlock_80(thr_id, endiandata);
	int warn = 0;

	do {
		int order = 0;

		// Hash with CUDA

		

		quark_blake512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id]);
	

		quark_bmw512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);

		quark_groestl512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);

		quark_skein512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);

		quark_jh512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);

		quark_keccak512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		

		x11_luffa512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);

		x11_cubehash512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);

		x11_shavite512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
				
		x11_simd512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);

		x11_echo512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);

		x13_hamsi512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);


		x13_fugue512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);

		x14_shabal512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);

		x15_whirlpool_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);

		x17_sha512_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id]); order++;

		x17_haval256_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id], 256); order++;


		*hashes_done = pdata[19] - first_nonce + throughput;

		work->nonces[0] = cuda_check_hash(thr_id, throughput, pdata[19], d_hash[thr_id]);
//#ifdef _DEBUG
//		uint32_t _ALIGN(64) dhash[8];
//		be32enc(&endiandata[19], pdata[19]);
//		x17_hash(dhash, endiandata);
//		applog_hash(dhash);
//		return -1;
//#endif
		if (work->nonces[0] != UINT32_MAX)
		{
			if (opt_benchmark) gpulog(LOG_BLUE, dev_id, "found");

			const uint32_t Htarg = ptarget[7];
			uint32_t _ALIGN(64) vhash[8];
			be32enc(&endiandata[19], work->nonces[0]);
			x17_hash(vhash, endiandata);

			if (vhash[7] <= Htarg && fulltest(vhash, ptarget)) {
				//gpulog(LOG_INFO, thr_id, "result info vhash 7 %08x,Htarg :%08x,all hash %08x%08x%08x%08x%08x%08x%08x%08x ,target: %08x%08x%08x%08x%08x%08x%08x%08x", vhash[7], Htarg, vhash[7], vhash[6], vhash[5], vhash[4], vhash[3], vhash[2], vhash[1], vhash[0], ptarget[7], ptarget[6], ptarget[5], ptarget[4], ptarget[3], ptarget[2], ptarget[1], ptarget[0]);
				//if (!opt_quiet)	gpulog(LOG_INFO, thr_id, "result for %08x validate on CPU! %s %s",
				//	work->nonces[0], algo_strings[algo80], hashOrder);

				work->valid_nonces = 1;
				work->nonces[1] = cuda_check_hash_suppl(thr_id, throughput, pdata[19], d_hash[thr_id], 1);
				work_set_target_ratio(work, vhash);
				if (work->nonces[1] != 0) {
					be32enc(&endiandata[19], work->nonces[1]);
					x17_hash(vhash, endiandata);
					bn_set_target_ratio(work, vhash, 1);
					work->valid_nonces++;
					pdata[19] = max(work->nonces[0], work->nonces[1]) + 1;
				} else {
					pdata[19] = work->nonces[0] + 1; // cursor
				}
#if 0
				gpulog(LOG_INFO, thr_id, "hash found with %s 80!", algo_strings[algo80]);

				algo80_tests[algo80] += work->valid_nonces;
				char oks64[128] = { 0 };
				char oks80[128] = { 0 };
				char fails[128] = { 0 };
				for (int a = 0; a < HASH_FUNC_COUNT; a++) {
					const char elem = hashOrder[a];
					const uint8_t algo64 = elem >= 'A' ? elem - 'A' + 10 : elem - '0';
					if (a > 0) algo64_tests[algo64] += work->valid_nonces;
					sprintf(&oks64[strlen(oks64)], "|%X:%2d", a, algo64_tests[a] < 100 ? algo64_tests[a] : 99);
					sprintf(&oks80[strlen(oks80)], "|%X:%2d", a, algo80_tests[a] < 100 ? algo80_tests[a] : 99);
					sprintf(&fails[strlen(fails)], "|%X:%2d", a, algo80_fails[a] < 100 ? algo80_fails[a] : 99);
				}
				applog(LOG_INFO, "K64: %s", oks64);
				applog(LOG_INFO, "K80: %s", oks80);
				applog(LOG_ERR,  "F80: %s", fails);
#endif
				return work->valid_nonces;
			}
			else if (vhash[7] > Htarg) {
				// x11+ coins could do some random error, but not on retry
				gpu_increment_reject(thr_id);
				if (!warn) {
					warn++;
					pdata[19] = work->nonces[0] + 1;
					continue;
				} else {
					//if (!opt_quiet)	{
					//	gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU! %s %s",
					//		work->nonces[0], algo_strings[algo80], hashOrder);
					//}
					warn = 0;
					return 0;
				}
			}
		}

		if ((uint64_t)throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}

		pdata[19] += throughput;

	} while (pdata[19] < max_nonce && !work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce;
	return 0;
}

// cleanup
extern "C" void free_x17(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipFree(d_hash[thr_id]);

	quark_blake512_cpu_free(thr_id);
	quark_groestl512_cpu_free(thr_id);
	x11_simd512_cpu_free(thr_id);
	x13_fugue512_cpu_free(thr_id);
	x16_fugue512_cpu_free(thr_id); // to merge with x13_fugue512 ?
	x15_whirlpool_cpu_free(thr_id);

	cuda_check_cpu_free(thr_id);

	hipDeviceSynchronize();
	init[thr_id] = false;
}
